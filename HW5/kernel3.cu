#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandeldevice(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int *output, int maxIterations, size_t pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int index = blockIdx.x*32*3 + threadIdx.x*3;
    
    for(int i=0 ; i<3 ; i++)
    {
      int thisY = index / 1600;
      int thisX = index % 1600;
      //int thisX = index - thisY*1600;
    
      float x = lowerX + thisX * stepX;
      float y = lowerY + thisY * stepY;

      int *head;

      head = (int*)((char*)output + thisY * pitch);

      //output[index]= mandeldevice(x, y, maxIterations);
      head[thisX] = mandeldevice(x, y, maxIterations);

      index++;
    }
}

//variables correspondence
//upperX -> x1
//upperY -> y1
//lowerX -> x0
//lowerY -> y0
//*img -> output
//resX -> width
//rexY -> height

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int img_size = resX * resY * sizeof(int);

    int *img_host;
    hipHostAlloc((void**)&img_host, img_size, hipHostMallocDefault);

    img_host = img;

    int *img_dev;

    size_t pitch;

    hipMallocPitch((void**)&img_dev, &pitch, resX*sizeof(int), resY);

    mandelKernel<<< 20000,32 >>>( lowerX, lowerY, stepX, stepY, img_dev, maxIterations, pitch);

    hipMemcpy2D(img_host, resX*sizeof(int), img_dev, pitch, resX*sizeof(int), resY, hipMemcpyDefault);


    hipFree(img_dev);
    //cudaFreeHost(img_host);
}
