#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandeldevice(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int *output, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int index = blockIdx.x*32 + threadIdx.x;
    
    int thisY = index / 1600;
    int thisX = index % 1600;
    //int thisX = index - thisY*1600;
    
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    output[index] = mandeldevice(x, y, maxIterations);


}

//variables correspondence
//upperX -> x1
//upperY -> y1
//lowerX -> x0
//lowerY -> y0
//*img -> output
//resX -> width
//rexY -> height

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int img_size = resX * resY * sizeof(int);

    int *img_host = (int*)malloc(img_size);

    img_host = img;

    int *img_dev;
    hipMalloc( (void**)&img_dev, img_size);

    mandelKernel<<< 60000,32 >>>( lowerX, lowerY, stepX, stepY, img_dev, maxIterations);

    hipMemcpy( img_host, img_dev, img_size, hipMemcpyDefault);

    //cudaMemcpy( img_host, img_dev, img_size, cudaMemcpyDeviceToHost );

    //cudaMemcpy( img, img_host, img_size, cudaMemcpyHostToHost);

    hipFree(img_dev);
}
