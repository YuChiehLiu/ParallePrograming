#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandeldevice(float c_re, float c_im, int count)
{
  //float z_re = c_re, z_im = c_im;
  int i=0;
  /*for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }*/

  /*float xtemp;

  while( (z_re * z_re + z_im * z_im <= 4.f) && (i<count) )
  {
    xtemp = z_re * z_re - z_im * z_im + c_re;
    z_im = 2.f*z_re*z_im + c_im;
    z_re = xtemp;
    i++;
  }*/

  float x = c_re;
  float y = c_im; 
  float x2 = x*x;
  float y2 = y*y;
  
  while( (x2+y2 <= 4.f) && (i<count) )
  {
    y = 2*x*y + c_im;
    x = x2-y2 + c_re;
    x2 = x*x;
    y2 = y*y;
    i++;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int *output, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int index = blockIdx.x*32 + threadIdx.x;
    
    int thisY = index / 1600;
    int thisX = index % 1600;
    //int thisX = index - thisY*1600;
    
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    output[index] = mandeldevice(x, y, maxIterations);


}

//variables correspondence
//upperX -> x1
//upperY -> y1
//lowerX -> x0
//lowerY -> y0
//*img -> output
//resX -> width
//rexY -> height

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int img_size = resX * resY * sizeof(int);

    int *img_dev;
    hipMalloc( (void**)&img_dev, img_size);

    mandelKernel<<< 60000,32 >>>( lowerX, lowerY, stepX, stepY, img_dev, maxIterations);

    hipMemcpy( img, img_dev, img_size, hipMemcpyDefault);

    //cudaMemcpy( img_host, img_dev, img_size, cudaMemcpyDeviceToHost );

    //cudaMemcpy( img, img_host, img_size, cudaMemcpyHostToHost);

    hipFree(img_dev);
}
