#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>

__device__ int mandeldevice(float c_re, float c_im, int count)
{
  //float z_re = c_re, z_im = c_im;
  /*for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }*/

  float xtemp;

  while( (z_re * z_re + z_im * z_im <= 4.f) && (i<count) )
  {
    xtemp = z_re * z_re - z_im * z_im + c_re;
    z_im = 2.f*z_re*z_im + c_im;
    z_re = xtemp;
    i++;
  }
  /*int i=0;
  float x = c_re;
  float y = c_im; 
  float x2 = x*x;
  float y2 = y*y;
  
  while( (x2+y2 <= 4.f) && (i<count) )
  {
    y = 2*x*y + c_im;
    x = x2-y2 + c_re;
    x2 = x*x;
    y2 = y*y;
    i++;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY,int *output, int maxIterations, size_t pitch)
{
    // To avoid error caused by the floating number, use the following pseudo code
    //
    int index = blockIdx.x*32 + threadIdx.x;
    
    int thisY = index / 1600;
    int thisX = index % 1600;
    //int thisX = index - thisY*1600;
    
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    int *head;
    head = (int*)((char*)output + thisY * pitch);

    //output[index]= mandeldevice(x, y, maxIterations);
    head[thisX] = mandeldevice(x, y, maxIterations);
}

//variables correspondence
//upperX -> x1
//upperY -> y1
//lowerX -> x0
//lowerY -> y0
//*img -> output
//resX -> width
//rexY -> height

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    /*int img_size = resX * resY * sizeof(int);

    int *img_host;
    hipHostAlloc((void**)&img_host, img_size, hipHostMallocDefault);

    img_host = img;*/

    int *img_dev;
    

    size_t pitch;

    hipMallocPitch((void**)&img_dev, &pitch, resX*sizeof(int), resY);

    mandelKernel<<< 60000,32 >>>( lowerX, lowerY, stepX, stepY, img_dev, maxIterations, pitch);

    hipMemcpy2D(img, resX*sizeof(int), img_dev, pitch, resX*sizeof(int), resY, hipMemcpyDefault);


    hipFree(img_dev);
    //hipHostFree(img_host);
}
