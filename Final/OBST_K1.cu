#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "CycleTimer.h"

using namespace std;

#define node 10000
#define key (node+1)
#define block_size 32
using namespace std;
void generate_data(int *p, int *q){
    srand(0);    
   
    for(int i=0; i<=node; i++){
        p[i] = rand() % 10;
        q[i] = rand() % 10;
    }
}
__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

__global__ void compute_w (int *w, int *p, int *q){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < (key) * ((key)+1) / 2){
        int temp = tid;
        int index_i = 1;
        for(int i=key; i>0; i--){
            if(tid - i < 0){
                break;
            }

            tid -= i;
            index_i++;
        }
        int index_j = index_i + tid -1 ;

        for(int k = index_i-1; k <= index_j; k++)
            w[index_i*(key) + index_j] += q[k];
        for(int k = index_i; k <= index_j; k++)
            w[index_i*(key) + index_j] += p[k];
    }
    return;
}

__global__ void find_min_root(int *min_s, int *s, int k){

    for(int index=0; index<(key-k-1); index++){
        int i = index+1;
        int temp = INT_MAX;
        for(int root = 0; root<(k+1); root++){
            int pos = (k==-1) ? 1 : (k+1);
            //printf("index %d of k = %d : %d \n",i ,k, min_s[index * pos + root]);
            if(min_s[index * pos + root] <= temp){
                 temp = min_s[index * pos + root];
            }
        }

        s[i * key + i+k] = temp;
        //printf("\n");
    }
    //printf("\n");

}

__global__ void compute_s (int *w, int *q,int *s, int k, int *min_s){

    //k = -1 to key-2
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(k == -1){
        if(tid >= key-k-1)
            return;
        else{
            s[(tid+1) * key + tid] = q[tid];
            //min_s[tid] = q[tid];
            return;
        }
    }
    else if(tid >= (key-k-1)*(k+1)){
        return;
    }
    else{
        int i = tid / (k+1) + 1;
        int r = i + tid % (k+1);

        min_s[tid] = s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k];

        //if(r == i){
        //    s[i*key+i+k] = INT_MAX;
        //}
        //__syncthreads();
        //int min = s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k];
        //
        ////lock
        //while(0 == atomicCAS(&mlock, 0, 1)){
        //    //critical section
        //    if(min < s[i*key+i+k]){
        //        s[i*key+i+k] = min;
        //    }

        //    //unlock
        //    atomicExch(&mlock, 0);
        //}

    }
    return;
}

int main(){
    //warm up gpu
    int grid_sizeW = ceil((double)(key) * ((double)(key) + 1) / 2.0 / (double)block_size);
    warm_up_gpu<<<grid_sizeW, block_size>>>();
    
    double start_time, end_time;

    //int w[key+1][key] = {0};
    //int s[key+1][key] = {0};
    int *w = (int *)malloc(sizeof(int) * (key+1) * key);
    int *s = (int *)malloc(sizeof(int) * (key+1) * key);
    int *p = (int*)malloc(key * sizeof(int));
    int *q = (int*)malloc(key * sizeof(int));
    generate_data(p, q);

    start_time = currentSeconds();
    int *w_d;
    hipMalloc(&w_d, sizeof(int) * (key+1) * (key));
    int *p_d, *q_d;
    hipMalloc(&p_d, sizeof(int) * (key));
    hipMalloc(&q_d, sizeof(int) * (key));
    hipMemcpy(p_d, p, sizeof(int) * (key), hipMemcpyDefault);
    hipMemcpy(q_d, q, sizeof(int) * (key), hipMemcpyDefault);

    compute_w<<<grid_sizeW, block_size>>>(w_d, p_d, q_d);
    hipDeviceSynchronize();
    hipMemcpy(w, w_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault );

    int *s_d;
    hipMalloc(&s_d, sizeof(int) * (key+1) * (key));
    hipMemcpy(s_d, s, sizeof(int) * (key+1) * (key), hipMemcpyDefault);

    for(int k=-1; k<=key-2 ; k++){
        int grid_sizeS = ceil( double((key-k-1) * (k+1)) / block_size);
        if(k == -1)
            grid_sizeS =  ceil( double(key-k-1) / block_size);

        int min_size = (k==-1) ? (key-k-1) : (key-k-1) * (k+1);

        int *min_s = (int *)malloc(sizeof(int) * min_size);
        memset(min_s, INT_MAX, min_size);
        int *min_s_d;
        hipMalloc(&min_s_d, sizeof(int) * min_size);
        hipMemcpy(min_s_d, min_s, sizeof(int) * min_size, hipMemcpyDefault);

        compute_s<<< grid_sizeS , block_size  >>>(w_d, q_d, s_d, k, min_s_d);
        hipDeviceSynchronize();
       
        if(k > -1){
            find_min_root<<<1, 1>>>(min_s_d, s_d, k);
        }

        hipDeviceSynchronize();

        free(min_s);
        hipFree(min_s_d);
    }

    hipMemcpy(s, s_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault);
    end_time = currentSeconds();
    
    //for(int i=1; i<=key; i++){
    //    for(int j=0; j<=key-1; j++){
    //        cout << w[i][j] << " ";
    //    }
    //    cout << endl;
    //}
    //
    //cout << endl;

    //for(int i=1; i<=key; i++){
    //    for(int j=0; j<=key-1; j++){
    //        cout << s[i][j] << " ";
    //    }
    //    cout << endl;
    //}

    cout << s[1 * key + node] << endl;

    double total_time = end_time - start_time;

    printf("\n[One kernel]:\t\t[%.3f] ms\n\n", total_time * 1000);

    free(s);
    free(w);
    free(p);
    free(q);
    hipFree(p_d);
    hipFree(q_d);
    hipFree(s_d);
    hipFree(w_d);
}
