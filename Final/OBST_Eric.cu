#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "CycleTimer.h"

#define node 150 
#define key (node+1)
#define block_size 32.0
using namespace std;
void generate_data(int *p, int *q){
    srand(0);    
   
    for(int i=0; i<=node; i++){
        p[i] = rand() % 10;
        q[i] = rand() % 10;
    }
}
__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

__global__ void compute_w (int *w, int *p, int *q){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < (key) * ((key)+1) / 2){
        int temp = tid;
        int index_i = 1;
        for(int i=key; i>0; i--){
            if(tid - i < 0){
                break;
            }

            tid -= i;
            index_i++;
        }
        int index_j = index_i + tid -1 ;

        for(int k = index_i-1; k <= index_j; k++)
            w[index_i*(key) + index_j] += q[k];
        for(int k = index_i; k <= index_j; k++)
            w[index_i*(key) + index_j] += p[k];
    }
    return;
}



__global__ void compute_s (int *w, int *q,int *s, int k){
    
    //k = -1 to key-2
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= key-k-1)
        return;
    int i = tid+1;
    if(k == -1){
        s[i*(key)+i+k] = q[i+k];
    }
    else{
        int r = i;
        s[i*key+i+k]  = s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k];
        for(r=i; r<=i+k;r++){
            if( s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k] < s[i*key + i+k])
                s[i*key+i+k] = s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k];
        }
    }
    //printf("\n\n");
    return;
}

int main(){
    //warm up gpu
    int grid_size_W = ceil((double)(key) * ((double)(key) + 1) / 2.0 / (double)block_size);
    warm_up_gpu<<<grid_size_W, block_size>>>();
    
    double start_time, end_time;

    int *w, *s;
    w = (int*)calloc((node+2) * (node+1), sizeof(int));
    s = (int*)calloc((node+2) * (node+1), sizeof(int));

    int *p = (int*)malloc(key * sizeof(int));
    int *q = (int*)malloc(key * sizeof(int));
    generate_data(p, q);

    //serial(p, q);

    start_time = currentSeconds();
    int *w_d;
    hipMalloc(&w_d, sizeof(int) * (key+1) * (key));
    int *p_d, *q_d;
    hipMalloc(&p_d, sizeof(int) * (key));
    hipMalloc(&q_d, sizeof(int) * (key));
    hipMemcpy(p_d, p, sizeof(int) * (key), hipMemcpyDefault);
    hipMemcpy(q_d, q, sizeof(int) * (key), hipMemcpyDefault);


    compute_w<<<grid_size_W, block_size>>>(w_d, p_d, q_d);
    hipDeviceSynchronize();

    hipMemcpy(w, w_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault );

    int *s_d;
    hipMalloc(&s_d, sizeof(int) * (key+1) * (key));
    

    for(int k=-1; k<=key-2 ; k++){
        //compute_s<<< 1,key-k-1 >>>(w_d, q_d, s_d, k);
        int grid_size_S = ceil((double)(key-k-1) / block_size );
        compute_s<<< grid_size_S, block_size >>>(w_d, q_d, s_d, k);
        hipDeviceSynchronize();
    }


    hipMemcpy(s, s_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault);
    end_time = currentSeconds();
    
    /*for(int i=1; i<=key; i++){
        for(int j=0; j<=key-1; j++){
            cout << s[i][j] << " ";
        }
        cout << endl;
    }*/

    cout << s[1*(node+1)+node] << endl;

    

    double total_time = end_time - start_time;

    printf("\n[eric]:\t\t[%.3f] ms\n\n", total_time * 1000);
}
