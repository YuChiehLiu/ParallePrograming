#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "CycleTimer.h"

#define node 100
#define key (node+1)
#define block_size 32
using namespace std;
void generate_data(int *p, int *q){
    srand(0);    
   
    for(int i=0; i<=node; i++){
        p[i] = rand() % 10;
        q[i] = rand() % 10;
    }
}
__global__ void warm_up_gpu(){
  unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;
  ib += ia + tid;
}

__global__ void compute_w (int *w, int *p, int *q){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if(tid < (key) * ((key)+1) / 2){
        int temp = tid;
        int index_i = 1;
        for(int i=key; i>0; i--){
            if(tid - i < 0){
                break;
            }

            tid -= i;
            index_i++;
        }
        int index_j = index_i + tid -1 ;

        for(int k = index_i-1; k <= index_j; k++)
            w[index_i*(key) + index_j] += q[k];
        for(int k = index_i; k <= index_j; k++)
            w[index_i*(key) + index_j] += p[k];
    }
    return;
}

__global__ void find_min_root(int i, int k, int *s,  int *w, int *min){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    int r = tid+i;

    //printf("%d \n r:%d \n s[%d][%d] : %d, s[%d][%d] : %d, w[%d][%d] : %d\n",tid , r, i, r-1, s[i*key+r-1], r+1, i+k, s[(r+1)*key+i+k], i, i+k, w[i*key+i+k]);
    min[tid]  = s[i*key+r-1] + s[(r+1)*key+i+k] +  w[i*key+i+k];

}



__global__ void compute_s (int *w, int *q,int *s, int k){
    
    //k = -1 to key-2
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int i = tid+1;
    if(k == -1){
        s[i*(key)+i+k] = q[i+k];
        //printf("s[%d][%d] = %d \n", i, i+k,s[i*(key)+i+k]);
    }
    else{
        int *min = (int *)malloc(sizeof(int) * (k+1));

        find_min_root<<<1,k+1>>>(i, k, s, w, min);
        
        hipDeviceSynchronize();

        //printf("\n");
        s[i*key+i+k] = INT_MAX;
        for(int index = 0; index<k+1; index++){
           // printf("%d ",  min[index]);
            if(min[index] <  s[i*key+i+k])
                s[i*key+i+k] = min[index];
        }
        //printf("s[%d][%d] = %d \n", i, i+k, s[i*(key)+i+k]);
    }
    //printf("\n\n");
    return;
}

int main(){
    //warm up gpu
    int grid_size = ceil((double)(key) * ((double)(key) + 1) / 2.0 / (double)block_size);
    warm_up_gpu<<<grid_size, block_size>>>();
    
    double start_time, end_time;

    int w[key+1][key] = {0};
    //int *w = (int*)calloc(key*(key+1) ,sizeof(int));
    int s[key+1][key] = {0};
    //int root[key][key];
    //int p[key] = {0,5,2,4,3};
    //int q[key] = {3,2,3,4,2};
    int *p = (int*)malloc(key * sizeof(int));
    int *q = (int*)malloc(key * sizeof(int));
    generate_data(p, q);

    //serial(p, q);

    start_time = currentSeconds();
    int *w_d;
    hipMalloc(&w_d, sizeof(int) * (key+1) * (key));
    int *p_d, *q_d;
    hipMalloc(&p_d, sizeof(int) * (key));
    hipMalloc(&q_d, sizeof(int) * (key));
    hipMemcpy(p_d, p, sizeof(int) * (key), hipMemcpyDefault);
    hipMemcpy(q_d, q, sizeof(int) * (key), hipMemcpyDefault);


    compute_w<<<grid_size, block_size>>>(w_d, p_d, q_d);
    hipDeviceSynchronize();

    hipMemcpy(w, w_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault );

    int *s_d;
    hipMalloc(&s_d, sizeof(int) * (key+1) * (key));
    

    for(int k=-1; k<=key-2 ; k++){
        //compute_s<<< 1,key-k-1 >>>(w_d, q_d, s_d, k);
        compute_s<<< key-k-1, 1 >>>(w_d, q_d, s_d, k);
        hipDeviceSynchronize();
    }

    hipMemcpy(s, s_d, sizeof(int) * (key+1) * (key),hipMemcpyDefault);
    hipDeviceSynchronize();

    end_time = currentSeconds();
    
    /*
    for(int i=1; i<=key; i++){
        for(int j=0; j<=key-1; j++){
            cout << s[i][j] << " ";
        }
        cout << endl;
    }
    */

    cout << s[1][key-1] << endl;

    

    double total_time = end_time - start_time;

    printf("\n[Dynamic Parallel]:\t\t[%.3f] ms\n\n", total_time * 1000);
}
