#include "hip/hip_runtime.h"
#include <iostream>
#include "CycleTimer.h"
#include <time.h>
#define node 2000 
#define key (node+1)
using namespace std;

void generate_data(int *p, int *q){
    
    srand(0);    
   
    for(int i=0; i<=node; i++){
        p[i] = rand() % 10;
        q[i] = rand() % 10;
    }
}

int main(){
    double start_time, end_time;
    start_time = currentSeconds();
    int *w, *s;
    w = (int*)calloc((node+2) * (node+1), sizeof(int));
    s = (int*)calloc((node+2) * (node+1), sizeof(int));

    /*
    int w[node+2][node+1] = {0};
    int s[node+2][node+1] = {0};
    */
    int *p = (int*)malloc(key * sizeof(int));
    int *q = (int*)malloc(key * sizeof(int));
    //int p[node+1] = {0,25,20,5,20,30};
    //int q[node+1] = {0};
    //int p[node+1] = {0, 5, 2, 4, 3};
    //int q[node+1] = {3, 2, 3, 4, 2};
    
    generate_data(p,q);
    for(int i=1; i<=node+1; i++){
        for(int j=0; j<=node; j++){
            if(i-j < 2){
                for(int k = i-1; k <= j; k++)
                    w[i*(node+1)+j] += q[k];
                for(int k = i; k <= j; k++)
                    w[i*(node+1)+j] += p[k];
            }
        }
    }

    for(int k=-1; k<= node-1; k++){
        for(int i=1;i<=node+1; i++){
            if(i+k > node)
                break;
            //s[i][i+k]
            int min = INT_MAX;
            if(k == -1)
                s[i*(node+1)+i+k] = q[i+k];
            else{
                for(int r = i; r<=i+k; r++){
                    s[i*(node+1)+i+k] = s[i*(node+1)+r-1] + s[(r+1)*(node+1)+i+k] +  w[i*(node+1)+i+k];
                    if(s[i*(node+1)+i+k] < min)
                        min = s[i*(node+1)+i+k];
                }
                s[i*(node+1)+i+k] = min;
            }
        }
    }
    end_time = currentSeconds();
    

    //for(int i=1; i<=node+1; i++){
    //    for(int j=0; j<=node; j++){
    //        cout << w[i][j] << " ";
    //    }
    //    cout << endl;
    //}

    //for(int i=1; i<=node+1; i++){
    //    for(int j=0; j<=node; j++){
    //        cout << w[i * (node+1) +j] << " ";
    //    }
    //    cout << endl;
    //}

    //cout<<endl;
    //    
    //for(int i=1; i<=node+1; i++){
    //    for(int j=0; j<=node; j++){
    //        cout << s[i * (node+1) +j] << " ";
    //    }
    //    cout << endl;
    //}
    
    

    cout << s[1*(node+1)+key-1] << endl;
    double total_time = end_time - start_time;

    printf("\n[Serial]:\t\t[%.3f] ms\n", total_time * 1000);
    free(w);
    free(s);
    free(q);
    free(p);
}
